
/* Henry Cooney <hacoo36@gmail.com> <Github: hacoo>
   AutoDockCuda: <https://github.com/hacoo/AutoDockCuda>

   start_CUDA.cu
   
   This file's job is to start the actual CUDA kernel. It
   should call needed kernels in order.

   This file also defines gpu memory pointers. Because CUDA
   is picky about how files are #included together, you should 
   #include all kernels after the gpu memory pointer decs.
   
*/

#ifndef _SUPPORT_H
#include "support.h"
#endif
#ifndef CUDA_HEADERS
#include "/pkgs/nvidia-cuda/5.5/include/hip/hip_runtime.h"
#include "/pkgs/nvidia-cuda/5.5/include/hip/hip_runtime.h"
#endif
#include "cuda_utils_host.h"

#include "gpu_variables.h"
#include "memory_layout.cuh"
#include "cuda_utils_gpu.cuh"



void start_CUDA_on_population(Population* this_pop, int ntors) {
  // Begins evaluation of Population on GPU. For now, this is a 
  // placeholder that will allow kernels to be tested, etc.
  

  int natoms = getNumAtoms((*this_pop)[0].mol);
  printf("Allocating %d atoms and %d torsions to GPU...\n", natoms, ntors);
    
  allocate_pop_to_gpu(*this_pop, ntors);
  
  dim3 dimBlock(natoms,1,1);
  dim3 dimGrid(2,1,1);

  printAutoDockMemoryKernel<<<dimGrid, dimBlock>>>();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) 
    printf("Error: %s\n", hipGetErrorString(err));

  printf("Done! \n");  
}

