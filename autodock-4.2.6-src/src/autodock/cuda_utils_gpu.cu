#include "hip/hip_runtime.h"

/* Henry Cooney <hacoo36@gmail.com> <Github: hacoo>
   AutoDockCuda: <https://github.com/hacoo/AutoDockCuda>

   cudat_utils_gpu.cu
   
   Includes utility functions for dealing with CUDA and autodock, on the GPU side.   
   includes functions for examing data on the GPU.

*/
#include "constants.h"
#include "typedefs.h"
#ifndef CUDA_HEADERS
#include "/pkgs/nvidia-cuda/5.5/include/hip/hip_runtime.h"
#include "/pkgs/nvidia-cuda/5.5/include/hip/hip_runtime.h"
#endif
#include <stdio.h>
#include <stdlib.h>
#ifndef CUDA_HEADERS
#include "/pkgs/nvidia-cuda/5.5/include/hip/hip_runtime.h"
#include "/pkgs/nvidia-cuda/5.5/include/hip/hip_runtime.h"
#endif
#ifndef CUDA_GPU_VARIABLES_H
#include "cuda_gpu_variables.cuh"
#endif


__global__
void printAutoDockMemoryKernel(int* natoms_dev) {
  // Testing / diagnostic kernel, prints 
  // contents of GPU memory to confirm they have been successfully
  // allocated.

  int idx = threadIdx.x;
  printf("Hello from thread %d! \n ", idx);
  if(idx == 0) {
    int natoms_local = *natoms_dev;
    // Since its a print function, threadIdx 0 will do all the work, very slowly.

    printf("Num atoms: %d \n ", natoms_local);
  }

}
