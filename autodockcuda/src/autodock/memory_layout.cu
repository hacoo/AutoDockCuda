#include "hip/hip_runtime.h"
#include "constants.h"
#include "typedefs.h"
#include <string.h>
#ifndef CUDA_HEADERS
#include "/pkgs/nvidia-cuda/5.5/include/hip/hip_runtime.h"
#include "/pkgs/nvidia-cuda/5.5/include/hip/hip_runtime.h"
#endif
#include <stdio.h>
#include <stdlib.h>
#ifndef _SUPPORT_H
#include "support.h"
#endif
#ifndef CUDA_UTILS_HOST_H
#include "cuda_utils_host.h"
#endif
#ifndef _STRUCTS_H
#include "structs.h"
#endif
#ifndef _AUTOCOMM
#include "autocomm.h"
#endif
#ifndef GPU_VARIABLES_H
#include "gpu_variables.h"
#endif
#ifndef CUDA_STRUCTS_H
#include "cuda_structs.h"
#endif



//const int ATOM_SIZE = (6 + MAX_TORS) * 3 * sizeof(Real);
//const int MOL_INDV_SIZE = (7 + MAX_TORS) * sizeof(Real) + MAX_ATOMS * ATOM_SIZE;


/////////////////////***********************************************/////////////////////
///****   THESE ARE THE UTILITY FUNCTIONS TO USE TO ACCESS DATA ON THE GPU    *****/////
/*
__device__ Real * getIndvAttribute(int idx) {
	//all data is packed into array in x,y,z,qw,qx,qy,qz, [torsion data], ......
	//returns the start address, move to next item by adding sizeof(Real)
	return globalReals + (idx * MOL_INDV_SIZE) * sizeof(Real);
	}*/
/*
__device__ Real * getTorsion(int indvIdx, int torsionIdx) {
	//all data is packed into array in x1,y1,z1,theta1, x2,y2, .....
	//returns the start address, move to next item by adding sizeof(Real)
	return globalReals + (indvIdx * MOL_INDV_SIZE + 7 + 4 * torsionIdx) * sizeof(Real);
	}*/
/*
__device__ char*  getAtom(int indvIdx, int atom) {
	//all data is packed into array in c11,c12,...c1MAX_CHARS, c21, c22, c23, ...
	//returns the start address, move to next item by adding sizeof(char)
  return (char*) (globalChars + (indvIdx * MAX_TORS * MAX_CHARS + atom * MAX_CHARS) * sizeof(char));
}
*/


bool allocate_pop_to_gpu(Population& pop_in, int ntors, CudaPtrs* ptrs) {
  
  int pop_size = pop_in.num_individuals();
  int state_size = 10 + ntors; // The total number of items in each STATE item
  // - 3 trans coords + 4 quat coords + 3 center coords + ntors torsions
  int i, ii;
  Molecule* first_mol = pop_in[0].mol; 
  State current_state; 

  int natoms = getNumAtoms(first_mol);  
  double* atom_crds = getAtomCrds(first_mol);
  double* torsions = getTorsions(first_mol, ntors);
  int* torsion_root_list = getTorsionRootList(first_mol, ntors); // List of torsion root atoms
  //int torlistsize = pop_size*MAX_TORS*MAX_ATOMS;
  //int torsion_root_list[torlistsize];
  char* atom_strings = getAtomStringArray(first_mol); // ragged array of atom strings
  
  double states[pop_size * state_size]; // flat array of individual states
  
  // Constant memory -- not implemented yet
  //  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(atom_crds_dev), atom_crds, sizeof(double)*natoms*SPACE));
  // gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(torsions_dev), torsions,  sizeof(double)*ntors*SPACE));
  /// gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(natoms_dev), &natoms, sizeof(int)));


  gpuErrchk(hipMalloc((void**) &(ptrs->natoms_dev),
		       sizeof(int)));
  gpuErrchk(hipMemcpy(ptrs->natoms_dev, &natoms, 
		       sizeof(int), hipMemcpyHostToDevice));

  gpuErrchk(hipMalloc((void**) &(ptrs->ntors_dev),
		       sizeof(int)));
  gpuErrchk(hipMemcpy(ptrs->ntors_dev, &ntors, 
		       sizeof(int), hipMemcpyHostToDevice));

  gpuErrchk(hipMalloc((void**) &(ptrs->state_size_dev),
		       sizeof(int)));
  gpuErrchk(hipMemcpy(ptrs->state_size_dev, &state_size, 
		       sizeof(int), hipMemcpyHostToDevice));



  gpuErrchk(hipMalloc((void**) &(ptrs->atom_crds_dev),
		       sizeof(double)*natoms*SPACE));
  gpuErrchk(hipMemcpy(ptrs->atom_crds_dev, atom_crds, 
		       sizeof(double)*natoms*SPACE, hipMemcpyHostToDevice));

  gpuErrchk(hipMalloc((void**) &(ptrs->torsions_dev),
		       sizeof(double)*ntors*SPACE));
  gpuErrchk(hipMemcpy(ptrs->torsions_dev, torsions, 
		       sizeof(double)*ntors*SPACE, hipMemcpyHostToDevice));
  
  gpuErrchk(hipMalloc((void**) &(ptrs->torsion_root_list_dev),
		       sizeof(int)*ntors*natoms));
  
  gpuErrchk(hipMemcpy(ptrs->torsion_root_list_dev, torsion_root_list, 
		       sizeof(int)*natoms*ntors, hipMemcpyHostToDevice));
  
/*
  for(i=0; i<pop_size; i++){
    for(int ii=0; ii<MAX_TORS; ++ii) {
        for (int iii=0; iii<MAX_ATOMS; ++iii) {
	        torsion_root_list[i*MAX_TORS*MAX_ATOMS+ii*MAX_ATOMS+iii] = 
                pop_in[i].mol->tlist[i][ii];
        }
    }
  }
*/
  gpuErrchk(hipMalloc((void**) &(ptrs->atom_strings_dev), 
		       sizeof(char)*natoms*MAX_CHARS));
  for(i=0; i<natoms; ++i){
    gpuErrchk(hipMemcpy(ptrs->atom_strings_dev+i*MAX_CHARS, atom_strings+i*MAX_CHARS,
	      MAX_CHARS,
	      hipMemcpyHostToDevice));
  }

  // TODO: Set constant memory addresses in ptr struct -- how do I make this work?
  //gpuErrchk(hipGetSymbolAddress((void**)&(ptrs->atom_crds_dev), atom_crds_dev));
  

  for (int i = 0; i < pop_size; ++i) {
    current_state = pop_in[i].phenotyp.make_state(ntors);
    // Translation:
    states[i*state_size] = current_state.T.x;
    states[i*state_size+1] = current_state.T.y;
    states[i*state_size+2] = current_state.T.z;
    // Quaternion
    states[i*state_size+3] = current_state.Q.w;
    states[i*state_size+4] = current_state.Q.x;
    states[i*state_size+5] = current_state.Q.y;
    states[i*state_size+6] = current_state.Q.z;
    // Center
    states[i*state_size+7] = current_state.Center.x;
    states[i*state_size+8] = current_state.Center.y;
    states[i*state_size+9] = current_state.Center.z;
    // Torsions
    for (ii=0; ii<ntors; ++ii) {
      states[i*state_size+10+ii] = current_state.tor[ii];
    }
  }

    
  // Allocate array of individual states -- it is a flat array containing 
  // the translation, rotation, and torsions of each individual
  gpuErrchk(hipMalloc((void**) &(ptrs->states_dev),
		       sizeof(double)*pop_size*state_size));
  gpuErrchk(hipMemcpy(ptrs->states_dev, states, 
		       sizeof(double)*state_size*pop_size, hipMemcpyHostToDevice));
  
  // Allocate array of inidividual atom coordinates -- also a flat array.
  // This array starts out initialized to 0 and is filled as calculations progress.
  gpuErrchk(hipMalloc((void**) &(ptrs->indiv_crds_dev),
		       sizeof(double)*pop_size*natoms*SPACE));
  gpuErrchk(hipMemset(ptrs->indiv_crds_dev, 0x00, 
		       sizeof(double)*pop_size*natoms*SPACE));



  // Allocate stuff related to eintcal_kernel
  
  Eval* peval = pop_in.evaluate;
  EnergyTables* p_etab = peval->get_energy_tables_ptr();
  
  Real* e_vdW_Hb_flattened = (Real*) malloc(sizeof(Real)*
						NEINT*MAX_ATOM_TYPES*MAX_ATOM_TYPES);
  
  int* is_hbond_flattened = (int*) malloc(sizeof(int)*
					 MAX_ATOM_TYPES*MAX_ATOM_TYPES);
  
  for(i=0; i<NEINT; ++i) {
    for(ii=0; ii<MAX_ATOM_TYPES; ++ii) {
      memcpy(e_vdW_Hb_flattened+(MAX_ATOM_TYPES*ii)+(MAX_ATOM_TYPES*MAX_ATOM_TYPES*i),
	     p_etab->e_vdW_Hb[i][ii],
	     MAX_ATOM_TYPES*sizeof(Real));
    }
  }
 
  gpuErrchk(hipMalloc((void**) &(ptrs->etab.e_vdW_Hb),
  		       sizeof(double)*NEINT*MAX_ATOM_TYPES*MAX_ATOM_TYPES));
  gpuErrchk(hipMemcpy(ptrs->etab.e_vdW_Hb,
		       e_vdW_Hb_flattened, 
		       sizeof(double)*NEINT*MAX_ATOM_TYPES*MAX_ATOM_TYPES,
		       hipMemcpyHostToDevice));

  gpuErrchk(hipMalloc((void**) &(ptrs->etab.sol_fn),
  		       sizeof(double)*NDIEL));
  gpuErrchk(hipMemcpy(ptrs->etab.sol_fn,
		       p_etab->sol_fn, 
		       sizeof(double)*NDIEL,
		       hipMemcpyHostToDevice));
  
  gpuErrchk(hipMalloc((void**) &(ptrs->etab.epsilon_fn),
  		       sizeof(double)*NDIEL));
  gpuErrchk(hipMemcpy(ptrs->etab.epsilon_fn,
		       p_etab->epsilon_fn, 
		       sizeof(double)*NDIEL,
		       hipMemcpyHostToDevice));

  gpuErrchk(hipMalloc((void**) &(ptrs->etab.r_epsilon_fn),
  		       sizeof(double)*NDIEL));
  gpuErrchk(hipMemcpy(ptrs->etab.r_epsilon_fn,
		       p_etab->r_epsilon_fn, 
		       sizeof(double)*NDIEL,
		       hipMemcpyHostToDevice));

  // Transfer the nonbond list
  int Nnb = peval->get_Nnb(); // number of nonbonds
  // The nonbondlist is a flat array of nonbond structs. It can be 
  // copied directly.
  printf("NNB: %d\n", Nnb);

  gpuErrchk(hipMalloc((void**) &(ptrs->nonbondlist),
  		       sizeof(NonbondParam)*Nnb));
  
  gpuErrchk(hipMemcpy(ptrs->nonbondlist,
		       peval->get_nonbondlist(), 
		       sizeof(NonbondParam)*Nnb,
		       hipMemcpyHostToDevice));
  
  // Transfer energy_component
  energy_component* pec = peval->get_energycomponent();
  if(pec) {
      gpuErrchk(hipMalloc((void**) &(ptrs->group_energy),
			   sizeof(EnergyComponent)));
      gpuErrchk(hipMemcpy(ptrs->group_energy,
			   pec, 
			   sizeof(EnergyComponent),
			   hipMemcpyHostToDevice)); 
  }
  else 
    ptrs->group_energy = NULL;
  
  gpuErrchk(hipMalloc((void**) &(ptrs->qsp_abs_charges),
		       sizeof(Real)*MAX_ATOMS));
  gpuErrchk(hipMemcpy(ptrs->qsp_abs_charges,
		       peval->get_qsp_abs_charge(), 
		       sizeof(EnergyComponent),
		       hipMemcpyHostToDevice)); 
  

  
  ptrs->Nnb = peval->get_Nnb();
  ptrs->B_calcIntElec = peval->get_B_calcIntElec();
  ptrs->B_include_1_4_interactions = peval->get_B_include_1_4_interactions();
  ptrs->B_use_non_bond_cutoff = peval->get_B_use_non_bond_cutoff();
  ptrs->B_have_flexible_residues = peval->get_B_have_flexible_residues();
  ptrs->scale_1_4 = peval->get_scale_1_4();

  // Nnb_array vector -- will be length 3 if have_flexible_residues is true, else is 
  // length 1.
  int nnb_array_length = ptrs->B_have_flexible_residues ? 3:1;
  int* p_nnb_array = peval->get_Nnb_array();
  gpuErrchk(hipMalloc((void**) &(ptrs->Nnb_array),
		       sizeof(int)*nnb_array_length));
  gpuErrchk(hipMemcpy(ptrs->Nnb_array,
		       p_nnb_array, 
		       sizeof(int)*nnb_array_length,
		       hipMemcpyHostToDevice));
  
 
  // Eintcal result vector
  gpuErrchk(hipMalloc((void**) &(ptrs->internal_energies_dev),
		       sizeof(double)*pop_size));
  gpuErrchk(hipMemset(ptrs->internal_energies_dev, 0.0,
		       sizeof(double)*pop_size));

  printf("Nnb: %d\n", ptrs->Nnb);
  printf("calcIntElec: %d\n", ptrs->B_calcIntElec);
  printf("B_include_1_4_interactions: %d\n", ptrs->B_include_1_4_interactions);
  printf("B_use_non_bond_cutoff: %d\n", ptrs->B_use_non_bond_cutoff);
  printf("B_have_flexible_residues: %d\n", ptrs->B_have_flexible_residues);
  printf("Scale_1_4: %f\n", ptrs->scale_1_4);
  printf("Nnb array: ");
  for(i=0; i<nnb_array_length; ++i) {
    printf(" %d ", p_nnb_array[i]);
  }
  printf("\n");
  printf("\n");
 

  
  free(atom_crds);
  free(atom_strings);
  free(torsions);
  free(torsion_root_list);
  
  return true;
}
